#include "hip/hip_runtime.h"
#include "gpu_algorithms.h"
#include "gpu_utils.h"
#include <cstdio>
#include <algorithm>
#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void calculate_fw_probs(
		double *d_fw_matrix,
		double *d_prob_matrix,
		inv_transition *d_inverse_neighbors,
		state_params *d_states,
		int num_of_states,
		int max_in_degree,
		int i,
		double emission)
{
	int l = threadIdx.x + blockIdx.x * blockDim.x;
	if (l < num_of_states){
		double em_prob = log(emission_probability(d_states[l].mean, d_states[l].stdv, emission));
		double sum = -INFINITY;
		int actual_neighbors = 0;
		for (int j = 0; j < max_in_degree; j++){
			inv_transition t = d_inverse_neighbors[l*max_in_degree + j];
			int k = t.state;
			if (k == -1) break;
			actual_neighbors ++;
			double gen_prob = log_mult(d_fw_matrix[(i-1) * num_of_states + k], t.prob);
			sum = log_sum(sum, log_mult(gen_prob, em_prob));
			d_prob_matrix[i * num_of_states * max_in_degree
				+ l * max_in_degree + j] = log_mult(gen_prob, em_prob);
		}
		d_fw_matrix[i * num_of_states + l] = sum;

		for (int j = actual_neighbors; j < max_in_degree; j++){
			int sh = i * num_of_states * max_in_degree + l * max_in_degree + j;
			d_prob_matrix[sh] = INFINITY;
		}
	}
}

__global__ void normalize(double *d_array, int len, int bound){
	int l = threadIdx.x + blockIdx.x * blockDim.x;
	if (l < bound){
		double sum = -INFINITY;
		for (int i = 0; i < len; i++){
			if (d_array[l * len + i] == INFINITY) break;
			sum = log_sum(d_array[l * len + i], sum);
		} 
		for (int i = 0; i < len; i++){
			if (d_array[l * len + i] == INFINITY) break;
			d_array[l * len + i] = log_div(d_array[l * len + i], sum);
		}
	}
}

__global__ void prefix_sum(double *d_prob_weights, int max_in_degree, int bound){
	int l = threadIdx.x + blockIdx.x * blockDim.x;
	if (l < bound){
		double sum = -INFINITY;
		for (int i = 0; i < max_in_degree; i++){
			if (d_prob_weights[l * max_in_degree + i] == INFINITY) break;
			sum = log_sum(d_prob_weights[l * max_in_degree + i], sum);
			d_prob_weights[l * max_in_degree + i] = sum;
		}
	}
}

void print_prob_matrix(double *prob_matrix, int seq_length, int num_of_states, int max_in_degree){
	for (int i = 0; i < seq_length; i++){
		for (int st = 0; st < num_of_states; st++){
			printf("[");
			for (int ne = 0; ne < max_in_degree; ne++){
				if (prob_matrix[i*num_of_states*max_in_degree + st*max_in_degree + ne] == INFINITY) break;
				printf("%.3f,", exp(prob_matrix[i*num_of_states*max_in_degree + st*max_in_degree + ne]));
			}
			printf("],");
		}
		printf("\n");
	}
}

void gpu_forward_matrix(
		std::vector<State> &states,
		std::vector<double> &event_sequence,
		inv_transition *d_inverse_neighbors,
		int num_of_states,
		int max_in_degree,
		double *d_prob_matrix,
		double *d_last_row_weights)
{

	int seq_length = event_sequence.size();
	LogNum init_transition_prob = LogNum(1.0/(double)num_of_states);

	double *fw_matrix = (double *)malloc(num_of_states * seq_length * sizeof(double));
	double *prob_matrix = (double *)malloc(num_of_states * max_in_degree * seq_length * sizeof(double));
	for (int i = 0; i < num_of_states; i++){
		fw_matrix [i] = (init_transition_prob * states[i].get_emission_probability(event_sequence[0])).exponent;
	}

	//convert states to struct type
	state_params *state_p = (state_params *)malloc(num_of_states * sizeof(state_params));
	for (int i = 0; i < num_of_states; i++){
		state_params s;
		s.mean = states[i].corrected_mean;
		s.stdv = states[i].corrected_stdv;
		state_p[i] = s;
	}

	for (int l = 0; l < num_of_states; l++){
		for (int j = 0; j < max_in_degree; j++){
			prob_matrix[l * max_in_degree + j] = -INFINITY;
		}
	}

	state_params *d_states;
	double *d_fw_matrix;
	hipMalloc((void **)&d_fw_matrix, seq_length * num_of_states * sizeof(double));
	hipMalloc((void **)&d_states, num_of_states * sizeof(state_params));

	hipMemcpy(d_fw_matrix, fw_matrix, num_of_states * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_prob_matrix, prob_matrix, num_of_states * max_in_degree * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_states, state_p, num_of_states * sizeof(state_params), hipMemcpyHostToDevice);

	int threads_per_block = 1024;
	int num_of_blocks = std::max((int)ceil((double)(num_of_states) / (double)threads_per_block), 1);

	hipEvent_t start_fwm, stop_fwm;
	hipEventCreate(&start_fwm);
	hipEventCreate(&stop_fwm);

	hipEventRecord(start_fwm);
	for (int i = 1; i < seq_length; i++){
		calculate_fw_probs<<<num_of_blocks, threads_per_block>>>(
				d_fw_matrix,
				d_prob_matrix,
				d_inverse_neighbors,
				d_states,
				num_of_states,
				max_in_degree,
				i,
				event_sequence[i]
			);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop_fwm);
	hipEventSynchronize(stop_fwm);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_fwm, stop_fwm);
	hipEventDestroy(start_fwm);
	hipEventDestroy(stop_fwm);
	
	threads_per_block = 1024;
	num_of_blocks = std::max((int)ceil((double)(num_of_states * seq_length) / (double)threads_per_block), 1);
	normalize<<<num_of_blocks, threads_per_block>>>(d_prob_matrix, max_in_degree, num_of_states * seq_length);
	hipDeviceSynchronize();

	hipFree(d_fw_matrix);
	hipFree(d_states);

	free(fw_matrix);
	free(prob_matrix);
	free(state_p);
}

__device__ int discrete_dist(double *weights, int max_in_degree, hiprandState *s){
	double val = log(hiprand_uniform(s));
	for (int i = 0; i < max_in_degree; i++){
		if (weights[i] == INFINITY) return (i - 1);
		if (log_less(val,weights[i])){
			return i;
		}
	}
	return max_in_degree -1;
}

__global__ void backtrack_sample(
								double *d_last_row_weights,
								double *d_prob_weights,
								inv_transition *d_inv_neighbors,
								int seq_length,
								int num_of_states,
								int max_in_degree,
								int num_of_samples,
								int *sample,
								int seed)
{
	unsigned int sample_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (sample_id < num_of_samples){

		hiprandState s;
		hiprand_init(seed, sample_id, 0, &s);

		int curr_state = discrete_dist(d_last_row_weights, num_of_states, &s);
		sample[sample_id * seq_length + seq_length - 1] = curr_state;
		int i = seq_length - 2;
		int rem_length = seq_length - 1;
		
		while (rem_length > 0){
			int shift = rem_length * num_of_states * max_in_degree + curr_state * max_in_degree;
			int next_state_id = discrete_dist(d_prob_weights + shift, max_in_degree, &s);
			curr_state = d_inv_neighbors[curr_state * max_in_degree + next_state_id].state;
			sample[sample_id * seq_length + i] = curr_state;
			rem_length--;
			i--;
		}
	}
}

std::vector<std::vector<int> > gpu_samples(
	int num_of_samples,
	std::vector<State> &states,
	std::vector<std::vector<std::pair<int, LogNum> > > &inverse_neighbors,
	int max_in_degree,
	std::vector<double>&event_sequence)
{

	hipEvent_t start_sampling, stop_sampling;
	hipEventCreate(&start_sampling);
	hipEventCreate(&stop_sampling);

	int num_of_states = states.size();
	int seq_length = event_sequence.size();

	//convert transitions to struct type
	inv_transition *inv_neighbors = (inv_transition *)malloc(num_of_states * max_in_degree *sizeof(inv_transition));
	for (int i = 0; i < num_of_states; i++){
		for (int j = 0; j < inverse_neighbors[i].size(); j++){
			inv_transition t;
			t.state = inverse_neighbors[i][j].first;
			t.prob = (inverse_neighbors[i][j].second).exponent;
			inv_neighbors[i * max_in_degree + j] = t;
		}
		for (int j = inverse_neighbors[i].size(); j < max_in_degree; j++){
			inv_transition t;
			t.state = -1;
			t.prob = INFINITY;
			inv_neighbors[i * max_in_degree + j] = t;
		}
	}

	double *d_prob_matrix;
	double *d_last_row_weights;
	inv_transition *d_inverse_neighbors;

	hipMalloc((void **)&d_prob_matrix, seq_length * num_of_states * max_in_degree * sizeof(double));
	hipMalloc((void **)&d_last_row_weights, num_of_states * sizeof(double));
	hipMalloc((void **)&d_inverse_neighbors, num_of_states * max_in_degree * sizeof(inv_transition));

	hipMemcpy(d_inverse_neighbors, inv_neighbors, num_of_states * max_in_degree *sizeof(inv_transition), hipMemcpyHostToDevice);

	gpu_forward_matrix(
		states,
		event_sequence,
		d_inverse_neighbors,
		num_of_states,
		max_in_degree,
		d_prob_matrix,
		d_last_row_weights);

	
	int threads_per_block = 1024;
	int num_of_blocks = std::max((int)ceil((double)(num_of_states * seq_length) / (double)threads_per_block), 1);
	prefix_sum<<<num_of_blocks, threads_per_block>>>(d_prob_matrix, max_in_degree, num_of_states * seq_length);
	hipDeviceSynchronize();

	normalize<<<1,1>>>(d_last_row_weights, num_of_states, 1);
	hipDeviceSynchronize();

	prefix_sum<<<1,1>>>(d_last_row_weights, num_of_states, 1);
	hipDeviceSynchronize();

	int *d_samples;
	hipMalloc((void **)&d_samples, seq_length * num_of_samples * sizeof(int));
	threads_per_block = 1024;
	num_of_blocks = std::max((int)ceil((double)(num_of_samples) / (double)threads_per_block), 1);
	backtrack_sample<<<num_of_blocks,threads_per_block>>>(
				d_last_row_weights,
				d_prob_matrix,
				d_inverse_neighbors,
				seq_length,
				num_of_states,
				max_in_degree,
				num_of_samples,
				d_samples,
				rand()
		);

	hipDeviceSynchronize();

	
	int *samples = (int *)malloc(seq_length * num_of_samples * sizeof(int));
	hipMemcpy(samples, d_samples, seq_length * num_of_samples * sizeof(int), hipMemcpyDeviceToHost);

	std::vector<std::vector<int> >r;
	for (int i = 0; i < num_of_samples; i++){
		std::vector<int>temp(samples + i * seq_length, samples + (i+1) * seq_length);
		r.push_back(temp);
	}

	hipFree(d_inverse_neighbors);
	hipFree(d_prob_matrix);
	hipFree(d_last_row_weights);
	hipFree(d_samples);
	free(inv_neighbors);
	free(samples);

	hipEventRecord(start_sampling);
	hipEventRecord(stop_sampling);
	hipEventSynchronize(stop_sampling);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_sampling, stop_sampling);
	hipEventDestroy(start_sampling);
	hipEventDestroy(stop_sampling);
	return r;
}