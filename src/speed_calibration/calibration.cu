
#include <hip/hip_runtime.h>


__global__ void kernel(double *arr, int len){
	double prev = 0.0;
	for (int i = 0; i < len; i++){
		if (i % 2){
			arr[i] = prev - i;
		}
		else{
			arr[i] = prev + i;
		}
		prev = arr[i];
	}
}

int launch_kernel(int n){
	double *d_arr;

	hipEvent_t start_fwm, stop_fwm;
	hipEventCreate(&start_fwm);
	hipEventCreate(&stop_fwm);
	hipEventRecord(start_fwm);

	hipMalloc((void **)&d_arr, n * sizeof(double));

	kernel<<<1,1>>>(d_arr, n);
	hipDeviceSynchronize();

	hipEventRecord(stop_fwm);
	hipEventSynchronize(stop_fwm);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_fwm, stop_fwm);
	hipEventDestroy(start_fwm);
	hipEventDestroy(stop_fwm);

	hipFree(d_arr);

	return (int)round(milliseconds);
}
